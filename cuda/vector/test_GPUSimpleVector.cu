#include "hip/hip_runtime.h"
//  author: Felice Pantaleo, CERN, 2018
#include "GPUSimpleVector.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <new>

__global__ void atomic_vector(GPU::SimpleVector<int> *foo) {
  auto index = threadIdx.x + blockIdx.x * blockDim.x;
  foo->push_back_ts(index);
}

int main() {
  auto maxN = 10000;
  GPU::SimpleVector<int> *obj_ptr = nullptr;
  GPU::SimpleVector<int> *d_obj_ptr = nullptr;
  GPU::SimpleVector<int> *tmp_obj_ptr = nullptr;
  int *data_ptr = nullptr;
  int *d_data_ptr = nullptr;

  bool success =
      hipHostMalloc(&obj_ptr, sizeof(GPU::SimpleVector<int>)) == hipSuccess &&
      hipHostMalloc(&data_ptr, maxN * sizeof(int)) == hipSuccess &&
      hipMalloc(&d_data_ptr, maxN * sizeof(int)) == hipSuccess;

  auto v = new (obj_ptr) GPU::SimpleVector<int>(maxN, data_ptr);

  hipHostMalloc(&tmp_obj_ptr, sizeof(GPU::SimpleVector<int>));
  new (tmp_obj_ptr) GPU::SimpleVector<int>(maxN, d_data_ptr);
  assert(tmp_obj_ptr->size() == 0);
  assert(tmp_obj_ptr->capacity() == static_cast<int>(maxN));

  success =
      success &&
      hipMalloc(&d_obj_ptr, sizeof(GPU::SimpleVector<int>)) == hipSuccess
      // ... and copy the object to the device.
      && hipMemcpy(d_obj_ptr, tmp_obj_ptr, sizeof(GPU::SimpleVector<int>),
                    hipMemcpyHostToDevice) == hipSuccess;

  int numBlocks = 5;
  int numThreadsPerBlock = 256;
  assert(success);
  atomic_vector<<<numBlocks, numThreadsPerBlock>>>(d_obj_ptr);

  hipMemcpy(obj_ptr, d_obj_ptr, sizeof(GPU::SimpleVector<int>),
             hipMemcpyDeviceToHost);

  assert(obj_ptr->size() == (numBlocks * numThreadsPerBlock < maxN
                                 ? numBlocks * numThreadsPerBlock
                                 : maxN));
  success = success and
            hipMemcpy(data_ptr, d_data_ptr, obj_ptr->size() * sizeof(int),
                       hipMemcpyDeviceToHost) == hipSuccess and
            hipHostFree(obj_ptr) == hipSuccess and
            hipHostFree(data_ptr) == hipSuccess and
            hipHostFree(tmp_obj_ptr) == hipSuccess and
            hipFree(d_data_ptr) == hipSuccess and
            hipFree(d_obj_ptr) == hipSuccess;
  assert(success);

  return 0;
}
